#include "hip/hip_runtime.h"
#include "CudaHelper.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define RUNTIME_CUDA
#include "wave_eq_func.h"

__device__
double* diaMulVec(
		unsigned int n,
		unsigned int diaDiags,
		const int* diaOffsets,
		const double* diaValues,
		const double* vec,
		double* res )
{
	// set result vector to zero
	for( unsigned int i = 0; i < n; ++i )
	{
		res[ i ] = 0.0;
	}
	for( unsigned int k = 0; k < diaDiags; ++k )
	{
		// determine first vector index involved in the diagonal multiplication
		unsigned int vecIndex = fmin( 0.0, diaOffsets[ k ] );
		// detemine the number of elements in the diagonal
		unsigned int diagSize = n - abs( diaOffsets[ k ] );
		for( unsigned int i = 0; i < diagSize; ++i )
		{
			res[ vecIndex + i ] += diaValues[ i ] * vec[ vecIndex + i ];
		}
		// increment values pointer to next diagonal
		diaValues += diagSize;
	}
	return res;
}

__device__
double* vecAddScaledVec(
		unsigned int n,
		double* u,
		double c,
		const double* v )
{
	for( unsigned int i = 0; i < n; ++i )
	{
		u[ i ] += c * v[ i ];
	}
	return u;
}

__device__
double* vecAddScaledVecs(
		unsigned int n,
		double c,
		const double* u,
		double d,
		const double* v,
		double* res )
{
	for( unsigned int i = 0; i < n; ++i )
	{
		res[ i ] = c * u[ i ] + d * v[ i ];
	}
	return res;
}

__global__
void mainKernel(
		unsigned int ip,
		unsigned int nsteps,
		double l2,
		unsigned int fdDiags,
		const int* fdOffsets,
		const double* fdValues,
		double* Z,
		double* W,
		double* U )
{
	// determine index of the first element of the subdomain
	unsigned int vecIndex = ( blockIdx.x * blockDim.x + threadIdx.x ) * ip;
	double* z = &Z[ vecIndex ];
	double* w = &W[ vecIndex ];
	double* u = &U[ vecIndex ];

	double a = 2 * ( 1 - l2 );
	double* swap;
	for( unsigned int i = 0; i < nsteps; ++i )
	{
		// u = M * z
		diaMulVec( ip, fdDiags, fdOffsets, fdValues, z, u );
		// u = u + a * z = M * z + a * z
		vecAddScaledVec( ip, u, a, z );
		// u = u + (-w) = M * z + a * z - w
		vecAddScaledVec( ip, u, -1.0, z );

		// shuffle buffers to avoid copying
		swap = w;
		w = z;
		z = u;
		u = swap;
	}
}

__global__
void initKernel(
		unsigned int ip,
		double L,
		double h,
		double dt,
		unsigned int fdDiags,
		const int* fdOffsets,
		const double* fdValues,
		double* Z,
		double* W,
		double* U )
{
	// determine index of the first element of the subdomain
	unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int vecIndex = id * ip;
	double* z = &Z[ vecIndex ];
	double* w = &W[ vecIndex ];
	double* u = &U[ vecIndex ];

	double x0 = h * ip * id / 2 - L;
	for( unsigned int i = 0; i < ip; ++i )
	{
		double x = x0 + i * h;
		w[ i ] = funu0( x );
		u[ i ] = funu1( x );
	}

	double a = 1 - dt / h * dt / h;
	// z = M * w
	diaMulVec( ip, fdDiags, fdOffsets, fdValues, w, z );
	// z = 0.5 * z + a * w = 0.5 * M * w + a * w
	vecAddScaledVecs( ip, 0.5, z, a, w, z );
	// z = z + dt * u
	vecAddScaledVec( ip, z, dt, u );
}

__global__
void syncKernel(
		unsigned int ip,
		double* Z,
		double* W )
{
	unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int n_1 = gridDim.x * blockDim.x - 1;
	unsigned int vecIndex = id * ip;
	double* z = &Z[ vecIndex ];
	double* w = &W[ vecIndex ];

	// determine indices of left and right neighbor subdomains, considering a
	// periodical continuation
	unsigned int leftNeighbor;
	unsigned int rightNeighbor;
	if( id == 0 )
	{
		leftNeighbor = n_1;
		rightNeighbor = 1;
	}
	else if( id == n_1 )
	{
		leftNeighbor = n_1 - 1;
		rightNeighbor = 0;
	}
	else
	{
		leftNeighbor = id - 1;
		rightNeighbor = id + 1;
	}

	// copy exact data from left neighbors
	double* nz = &Z[ leftNeighbor * ip ];
	double* nw = &W[ leftNeighbor * ip ];
	for( unsigned int i = 0; i < ip / 4; ++i )
	{
		z[ i ] = nz[ ip / 2 + i ];
		w[ i ] = nw[ ip / 2 + i ];
	}
	// copy exact data from right neighbors
	nz = &Z[ rightNeighbor * ip ];
	nw = &W[ rightNeighbor * ip ];
	for( unsigned int i = 0; i < ip / 4; ++i )
	{
		z[ ip * 3 / 4 + i ] = nz[ ip / 4 + i ];
		w[ ip * 3 / 4 + i ] = nw[ ip / 4 + i ];
	}
}

__global__
void reassociationKernel(
		unsigned int ip,
		const double* Z,
		double* S )
{
	unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int vecIndex = id * ip;
	const double* z = &Z[ vecIndex ];
	double* s = &S[ vecIndex / 2 ];

	// copy left half of the subdomain solution
	for( unsigned int i = 0; i < ip / 2; ++i )
	{
		s[ i ] = z[ i ];
	}
}

void CudaHelper::callMainKernel(
		unsigned int blocks,
		unsigned int threads,
		unsigned int ip,
		unsigned int nsteps,
		double l2,
		unsigned int fdDiags,
		const int* fdOffsets,
		const double* fdValues,
		double* Z,
		double* W,
		double* U )
{
	mainKernel<<<blocks, threads>>>(
			ip,
			nsteps,
			l2,
			fdDiags,
			fdOffsets,
			fdValues,
			Z,
			W,
			U );
}

void CudaHelper::callInitKernel(
		unsigned int blocks,
		unsigned int threads,
		unsigned int ip,
		double L,
		double h,
		double dt,
		unsigned int fdDiags,
		const int* fdOffsets,
		const double* fdValues,
		double* Z,
		double* W,
		double* U )
{
	initKernel<<<blocks, threads>>>(
			ip,
			L,
			h,
			dt,
			fdDiags,
			fdOffsets,
			fdValues,
			Z,
			W,
			U );
}

void CudaHelper::callSyncKernel(
		unsigned int blocks,
		unsigned int threads,
		unsigned int ip,
		double* Z,
		double* W )
{
	syncKernel<<<blocks, threads>>>( ip, Z, W );
}

void CudaHelper::callReassociationKernel(
		unsigned int blocks,
		unsigned int threads,
		unsigned int ip,
		const double* Z,
		double* S )
{
	reassociationKernel<<<blocks, threads>>>( ip, Z, S );
}

template<typename T>
T* CudaHelper::allocDevMem( size_t numElem )
{
	T* mem;
	hipMalloc( &mem, numElem * sizeof( T ) );
	return mem;
}

template<typename T>
void CudaHelper::freeDevMem( T* mem )
{
	hipFree( mem );
}

template<typename T>
void CudaHelper::copyHostToDevMem( const T* hostMem, T* devMem, size_t numElem )
{
	hipMemcpy(
			devMem,
			hostMem,
			numElem * sizeof( T ),
			hipMemcpyHostToDevice );
}

template<typename T>
void CudaHelper::copyDevToHostMem( const T* devMem, T* hostMem, size_t numElem )
{
	hipMemcpy(
			hostMem,
			devMem,
			numElem * sizeof( T ),
			hipMemcpyDeviceToHost );
}

/* explicitly define template instantiations used in the program (otherwise they
   won't be compiled, as nvcc doesn't know about their use, since all but this
   class is compiled by g++ */

template int* CudaHelper::allocDevMem<int>( size_t );
template double* CudaHelper::allocDevMem<double>( size_t );

template void CudaHelper::freeDevMem<int>( int* );
template void CudaHelper::freeDevMem<double>( double* );

template void CudaHelper::copyHostToDevMem<int>( const int*, int*, size_t );
template void CudaHelper::copyHostToDevMem<double>( const double*, double*, size_t );

template void CudaHelper::copyDevToHostMem<int>( const int*, int*, size_t );
template void CudaHelper::copyDevToHostMem<double>( const double*, double*, size_t );

