#include "hip/hip_runtime.h"
#include "CudaSolver.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

__global__
void kernel(
		unsigned int ip,
		unsigned int nsteps,
		double a,
		unsigned int fdDiags,
		int* fdOffsets,
		double* fdValues,
		double* Z,
		double* W,
		double* U )
{
}

void CudaSolver::callKernel(
		unsigned int blocks,
		unsigned int threads,
		unsigned int ip,
		unsigned int nsteps,
		double a,
		unsigned int fdDiags,
		int* fdOffsets,
		double* fdValues,
		double* Z,
		double* W,
		double* U )
{
	kernel<<<blocks, threads>>>(
			ip,
			nsteps,
			a,
			fdDiags,
			fdOffsets,
			fdValues,
			Z,
			W,
			U );
}
